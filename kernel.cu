#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include "md5_gpu.cu"
#include "md5.h"
#include <fstream>
#include <ctime>

#define CONST_FILE_LENGTH 20000
#define CONST_WORD_MAX_LENGTH 12

using namespace std;

 /* Global variables */
uint8_t g_wordLength;
char g_cracked[CONST_WORD_MAX_LENGTH];
string g_words[CONST_FILE_LENGTH];
char g_charArray[CONST_FILE_LENGTH][CONST_WORD_MAX_LENGTH];

__device__ char g_deviceCracked[CONST_WORD_MAX_LENGTH];
__device__ char g_deviceCharArray[CONST_FILE_LENGTH][CONST_WORD_MAX_LENGTH];

__global__ void md5Crack(uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	if (i < CONST_FILE_LENGTH)
	{
		/* Thread variables */
		uint32_t threadHash01, threadHash02, threadHash03, threadHash04;
		uint32_t threadWordLength = sizeof(g_deviceCharArray[i]) / sizeof(char);
		char threadTextWord[CONST_WORD_MAX_LENGTH];

		uint32_t charsNum = 0;
		for (uint32_t j = 0; j < threadWordLength; j++) {
			if (g_deviceCharArray[i][j] != 0) charsNum++;
			threadTextWord[j] = g_deviceCharArray[i][j];
		}

		md5Hash((unsigned char*)threadTextWord, charsNum, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

		if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04) {
			memcpy(g_deviceCracked, threadTextWord, threadWordLength);
		}
	}
	__syncthreads();
}

int findHashCPU(string input[CONST_FILE_LENGTH], string inputHash)
{
	int found = 0;
	for (unsigned int i = 0; i < CONST_FILE_LENGTH; i = i + 1)
	{
		string data = input[i];
		string data_hex_digest;

		MD5 hash;
		if (inputHash == hash(data)) {
			found = i;
		}
	}
	return found;
}

int main() {
	/* password hash to find: 06041992, 20000*/
	char passwordHash[33] = "f439a66cf295371bd7b674ef571e8815";

	/* read text file to array*/
	string wordsArray[CONST_FILE_LENGTH];
	ifstream file("passwords.txt");
	if (file.is_open()) for (int i = 0; i < CONST_FILE_LENGTH; ++i) file >> wordsArray[i];

	/*-------------------------------------------------------------------------------------------*/
	/*CPU PART*/
	std::clock_t c_start = std::clock();

	int index = findHashCPU(wordsArray, passwordHash);

	std::clock_t c_end = std::clock();

	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;

	std::cout << "CPU bruteforce:\n\t\cracked word: " << wordsArray[index] << std::endl;
	std::cout << "\ttime used: " << time_elapsed_ms << " ms\n";



	/*-------------------------------------------------------------------------------------------*/
	/*GPU PART*/
	/* convert string array to char[][]*/
	memset(g_charArray, 0, CONST_FILE_LENGTH);

	for (int i = 0; i < CONST_FILE_LENGTH; i++) {
		for (int j = 0; j < wordsArray[i].length(); j++) {
			g_charArray[i][j] = wordsArray[i][j];
		}
	}

	/* variable for hash stored as u32 integers */
	uint32_t md5Hash[4];

	/* parse hash to u32 integer */
	for (uint8_t i = 0; i < 4; i++) {
		char tmp[16];
		strncpy(tmp, passwordHash + i * 8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}

	/* fill memory */
	memset(g_cracked, 0, CONST_WORD_MAX_LENGTH);
	g_wordLength = CONST_WORD_MAX_LENGTH;

	/* copy to device */
	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_MAX_LENGTH, 0, hipMemcpyHostToDevice);
	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCharArray), g_charArray, sizeof(char) * CONST_FILE_LENGTH * CONST_WORD_MAX_LENGTH, 0, hipMemcpyHostToDevice);

	hipEvent_t clockBegin;
	hipEvent_t clockLast;

	hipEventCreate(&clockBegin);
	hipEventCreate(&clockLast);
	hipEventRecord(clockBegin, 0);

	md5Crack <<< 40, 500 >>> (md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

	/* Copy result */
	hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_MAX_LENGTH, 0, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventRecord(clockLast, 0);
	hipEventSynchronize(clockLast);
	hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

	hipEventDestroy(clockBegin);
	hipEventDestroy(clockLast);

	std::cout << "\nGPU bruteforce:\n\tcracked word: " << g_cracked << std::endl;
	std::cout << "\tcomputation time: " << milliseconds << " ms" << std::endl;
}