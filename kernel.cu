#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include "md5_gpu.cu"
#include "md5.h"
#include <fstream>

#define CONST_FILE_LENGTH 1000
#define CONST_WORD_LENGTH_KRISTOF 5

using namespace std;

 /* Global variables */
uint8_t g_wordLength;
char g_cracked[CONST_WORD_LENGTH_KRISTOF];
string g_words[CONST_FILE_LENGTH];

__device__ char g_deviceCracked[CONST_WORD_LENGTH_KRISTOF];
//__device__ string g_deviceWords[CONST_FILE_LENGTH];

__global__ void md5Crack(uint8_t wordLength, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04) {

	/* Thread variables */
	char threadTextWord[6] = "teszt";
	uint8_t threadWordLength;
	uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

	/* Copy everything to local memory */
	memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));

	md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

	if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04) {
		memcpy(g_deviceCracked, threadTextWord, threadWordLength);
	}
}

int findHashCPU(string input[CONST_FILE_LENGTH], string inputHash)
{
	for (unsigned int i = 0; i < CONST_FILE_LENGTH; i = i + 1)
	{
		string data = input[i];
		string data_hex_digest;

		MD5 hash;
		if (inputHash == hash(data)) {
			return i;
		}
	}
	return 0;
}

int main() {
	/* password hash to find: andre*/
	char passwordHash[33] = "19984dcaea13176bbb694f62ba6b5b35";

	/* read text file to array*/
	string wordsArray[CONST_FILE_LENGTH];
	ifstream file("passwords.txt");
	if (file.is_open()) for (int i = 0; i < CONST_FILE_LENGTH; ++i) file >> wordsArray[i];

	/**/
	int index = findHashCPU(wordsArray, passwordHash);
	std::cout << "found index: " << index << std::endl;

	/* variable for hash stored as u32 integers */
	uint32_t md5Hash[4];

	/* parse hash to u32 integer */
	for (uint8_t i = 0; i < 4; i++) {
		char tmp[16];
		strncpy(tmp, passwordHash + i * 8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}

	/* fill memory */
	memset(g_cracked, 0, CONST_WORD_LENGTH_KRISTOF);
	//memset(g_deviceWords, 0, CONST_FILE_LENGTH);
	g_wordLength = CONST_WORD_LENGTH_KRISTOF;

	/* copy to device */
	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyHostToDevice);

	hipEvent_t clockBegin;
	hipEvent_t clockLast;

	hipEventCreate(&clockBegin);
	hipEventCreate(&clockLast);
	hipEventRecord(clockBegin, 0);

	md5Crack <<< 1,1 >>> (g_wordLength, md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

	/* Copy result */
	hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventRecord(clockLast, 0);
	hipEventSynchronize(clockLast);
	hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

	std::cout << "computation time: " << milliseconds << " ms" << std::endl;

	hipEventDestroy(clockBegin);
	hipEventDestroy(clockLast);

	std::cout << "cracked word: " << g_cracked << std::endl;
}