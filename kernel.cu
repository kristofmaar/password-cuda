#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include "md5.cu"

#define CONST_WORD_LENGTH_KRISTOF 5


 /* Global variables */
uint8_t g_wordLength;
char g_cracked[CONST_WORD_LENGTH_KRISTOF];

__device__ char g_deviceCracked[CONST_WORD_LENGTH_KRISTOF];

__global__ void md5Crack(uint8_t wordLength, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04) {

	/* Thread variables */
	char threadTextWord[6] = "teszt";
	uint8_t threadWordLength;
	uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

	/* Copy everything to local memory */
	memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));

	md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

	if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04) {
		memcpy(g_deviceCracked, threadTextWord, threadWordLength);
	}
}

int main() {
	/* hash stored as u32 integers */
	uint32_t md5Hash[4];

	/* parse hash to u32 integer */
	for (uint8_t i = 0; i < 4; i++) {
		char tmp[16];
		strncpy(tmp, "6c90aa3760658846a86a263a4e92630e" + i * 8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}

	/* Fill memory */
	memset(g_cracked, 0, CONST_WORD_LENGTH_KRISTOF);
	g_wordLength = CONST_WORD_LENGTH_KRISTOF;

	/* copy to device */
	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyHostToDevice);

	md5Crack <<< 1,1 >>> (g_wordLength, md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

	/* Copy result */
	hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyDeviceToHost);

	std::cout << "Notice: cracked " << g_cracked << std::endl;
}