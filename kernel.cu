#include "hip/hip_runtime.h"
#include <stdio.h>
#include <iostream>
#include <time.h>
#include <string.h>
#include <stdlib.h>
#include <stdint.h>
#include <sstream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/device_functions.h>
#include "md5_gpu.cu"
#include "md5.h"
#include <fstream>
#include <ctime>

#define CONST_FILE_LENGTH 10000
#define CONST_WORD_LENGTH_KRISTOF 5

using namespace std;

 /* Global variables */
uint8_t g_wordLength;
char g_cracked[CONST_WORD_LENGTH_KRISTOF];
string g_words[CONST_FILE_LENGTH];

__device__ char g_deviceCracked[CONST_WORD_LENGTH_KRISTOF];
//__device__ string g_deviceWords[CONST_FILE_LENGTH];

__global__ void md5Crack(uint8_t wordLength, uint32_t hash01, uint32_t hash02, uint32_t hash03, uint32_t hash04) {

	/* Thread variables */
	char threadTextWord[6] = "teszt";
	uint8_t threadWordLength;
	uint32_t threadHash01, threadHash02, threadHash03, threadHash04;

	/* Copy everything to local memory */
	memcpy(&threadWordLength, &wordLength, sizeof(uint8_t));

	md5Hash((unsigned char*)threadTextWord, threadWordLength, &threadHash01, &threadHash02, &threadHash03, &threadHash04);

	if (threadHash01 == hash01 && threadHash02 == hash02 && threadHash03 == hash03 && threadHash04 == hash04) {
		memcpy(g_deviceCracked, threadTextWord, threadWordLength);
	}
}

int findHashCPU(string input[CONST_FILE_LENGTH], string inputHash)
{
	int found = 0;
	for (unsigned int i = 0; i < CONST_FILE_LENGTH; i = i + 1)
	{
		string data = input[i];
		string data_hex_digest;

		MD5 hash;
		if (inputHash == hash(data)) {
			found = i;
		}
	}
	return found;
}

int main() {
	/* password hash to find: cattle*/
	char passwordHash[33] = "bd5f4b0419caa97dd2f9b4d3238ff92f";

	/* read text file to array*/
	string wordsArray[CONST_FILE_LENGTH];
	ifstream file("passwords.txt");
	if (file.is_open()) for (int i = 0; i < CONST_FILE_LENGTH; ++i) file >> wordsArray[i];

	/*CPU hash finder*/
	std::clock_t c_start = std::clock();

	int index = findHashCPU(wordsArray, passwordHash);

	std::clock_t c_end = std::clock();

	double time_elapsed_ms = 1000.0 * (c_end - c_start) / CLOCKS_PER_SEC;

	std::cout << "CPU bruteforce:\n\cracked word: " << wordsArray[index] << std::endl;
	std::cout << "\ttime used: " << time_elapsed_ms << " ms\n";

	/* variable for hash stored as u32 integers */
	uint32_t md5Hash[4];

	/* parse hash to u32 integer */
	for (uint8_t i = 0; i < 4; i++) {
		char tmp[16];
		strncpy(tmp, passwordHash + i * 8, 8);
		sscanf(tmp, "%x", &md5Hash[i]);
		md5Hash[i] = (md5Hash[i] & 0xFF000000) >> 24 | (md5Hash[i] & 0x00FF0000) >> 8 | (md5Hash[i] & 0x0000FF00) << 8 | (md5Hash[i] & 0x000000FF) << 24;
	}

	/* fill memory */
	memset(g_cracked, 0, CONST_WORD_LENGTH_KRISTOF);
	//memset(g_deviceWords, 0, CONST_FILE_LENGTH);
	g_wordLength = CONST_WORD_LENGTH_KRISTOF;

	/* copy to device */
	hipMemcpyToSymbol(HIP_SYMBOL(g_deviceCracked), g_cracked, sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyHostToDevice);

	hipEvent_t clockBegin;
	hipEvent_t clockLast;

	hipEventCreate(&clockBegin);
	hipEventCreate(&clockLast);
	hipEventRecord(clockBegin, 0);

	md5Crack <<< 1,1 >>> (g_wordLength, md5Hash[0], md5Hash[1], md5Hash[2], md5Hash[3]);

	/* Copy result */
	hipMemcpyFromSymbol(g_cracked, HIP_SYMBOL(g_deviceCracked), sizeof(uint8_t) * CONST_WORD_LENGTH_KRISTOF, 0, hipMemcpyDeviceToHost);

	float milliseconds = 0;
	hipEventRecord(clockLast, 0);
	hipEventSynchronize(clockLast);
	hipEventElapsedTime(&milliseconds, clockBegin, clockLast);

	hipEventDestroy(clockBegin);
	hipEventDestroy(clockLast);

	std::cout << "\nGPU bruteforce:\n\tcracked word: " << g_cracked << std::endl;
	std::cout << "\tcomputation time: " << milliseconds << " ms" << std::endl;
}